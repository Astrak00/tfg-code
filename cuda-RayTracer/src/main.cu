#include "hip/hip_runtime.h"
#include "rtweekend.cuh"

#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <string_view>
#include <vector>

#include <hip/hip_runtime.h>

// CUDA kernel to render the scene
__global__ void render_kernel(vec3 * fb, int max_x, int max_y, camera * cam, hittable * world,
                              hiprandState * rand_state) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;

  if ((i >= max_x) || (j >= max_y)) return;

  int pixel_index = j * max_x + i;

  hiprandState local_rand_state = rand_state[pixel_index];

  color pixel_color(0, 0, 0);
  for (int sample = 0; sample < cam->samples_per_pixel; sample++) {
    ray const r = cam->get_ray(i, j, &local_rand_state);
    pixel_color += cam->ray_color(r, cam->max_depth, *world, &local_rand_state);
  }
  fb[pixel_index] = pixel_color;

  rand_state[pixel_index] = local_rand_state;
}

int main(int argc, char * argv[]) {
  // Default path for sphere data
  std::string sphere_data_path = "sphere_data.txt";
  std::string output_ppm_path  = "cuda_spheres.ppm";

  // Process command-line arguments
  for (int i = 1; i < argc; i++) {
    std::string_view arg(argv[i]);

    if (arg == "--path") {
      if (i + 1 < argc) {
        sphere_data_path = argv[i + 1];
        i++;
      } else {
        std::cerr << "Error: --path requires a value\n";
        return 1;
      }
    } else if (arg == "--output") {
      if (i + 1 < argc) {
        output_ppm_path = argv[i + 1];
        i++;
      } else {
        std::cerr << "Error: --output requires a value\n";
        return 1;
      }
    } else if (arg == "--help" || arg == "-h") {
      std::cout << "Usage: " << argv[0]
                << " [--path <sphere_data_path>] [--output <output_ppm_path>]\n";
      std::cout << "Default sphere data path: " << sphere_data_path << "\n";
      std::cout << "Default output PPM path: " << output_ppm_path << "\n";
      return 0;
    } else {
      std::cerr << "Error: Unknown argument: " << arg << "\n";
      std::cerr << "Use --help for usage information\n";
      return 1;
    }
  }

  // World and camera setup
  std::vector<hittable *> hittable_objects;
  camera cam;

  // Default camera settings
  cam.aspect_ratio      = 16.0 / 9.0;
  cam.image_width       = 800;
  cam.samples_per_pixel = 50;
  cam.max_depth         = 50;
  cam.vfov              = 20;
  cam.lookfrom          = point3(13, 2, 3);
  cam.lookat            = point3(0, 0, 0);
  cam.vup               = vec3(0, 1, 0);
  cam.defocus_angle     = 0.6;
  cam.focus_dist        = 10.0;

  std::ifstream infile(sphere_data_path);
  if (!infile.is_open()) {
    std::cerr << "Could not open file: " << sphere_data_path << std::endl;
    return 1;
  }
  std::string line;
  while (std::getline(infile, line)) {
    if (line.empty() || line[0] == '#') { continue; }
    std::istringstream iss(line);

    std::string first_token;
    iss >> first_token;

    if (first_token == "c") {
      std::string param_name;
      iss >> param_name;

      if (param_name == "ratio") {
        double width, height;
        if (iss >> width >> height) { cam.aspect_ratio = width / height; }
      } else if (param_name == "width") {
        int width;
        if (iss >> width) { cam.image_width = width; }
      } else if (param_name == "samplesPerPixel") {
        int samples;
        if (iss >> samples) { cam.samples_per_pixel = samples; }
      } else if (param_name == "maxDepth") {
        int depth;
        if (iss >> depth) { cam.max_depth = depth; }
      } else if (param_name == "vfov") {
        double vfov;
        if (iss >> vfov) { cam.vfov = vfov; }
      } else if (param_name == "lookFrom") {
        double x, y, z;
        if (iss >> x >> y >> z) { cam.lookfrom = point3(x, y, z); }
      } else if (param_name == "lookAt") {
        double x, y, z;
        if (iss >> x >> y >> z) { cam.lookat = point3(x, y, z); }
      } else if (param_name == "vup") {
        double x, y, z;
        if (iss >> x >> y >> z) { cam.vup = vec3(0, 1, 0); }
      } else if (param_name == "defocusAngle") {
        double angle;
        if (iss >> angle) { cam.defocus_angle = angle; }
      } else if (param_name == "focusDist") {
        double dist;
        if (iss >> dist) { cam.focus_dist = dist; }
      }
      continue;
    }

    iss.clear();
    iss.seekg(0);

    double x, y, z, radius;
    std::string material_type;
    if (!(iss >> x >> y >> z >> radius >> material_type)) { continue; }

    material * sphere_material;
    if (material_type == "lambertian") {
      double r, g, b;
      if (!(iss >> r >> g >> b)) { continue; }
      sphere_material = new lambertian(color(r, g, b));
    } else if (material_type == "metal") {
      double r, g, b, fuzz;
      if (!(iss >> r >> g >> b >> fuzz)) { continue; }
      sphere_material = new metal(color(r, g, b), fuzz);
    } else if (material_type == "dielectric") {
      double index;
      if (!(iss >> index)) { continue; }
      sphere_material = new dielectric(index);
    } else {
      continue;
    }
    point3 const center(x, y, z);
    hittable_objects.push_back(new sphere(center, radius, sphere_material));
  }
  std::cout << "Loaded world from " << sphere_data_path << "\n";
  infile.close();

  cam.initialize();

  // Allocate memory on the device
  int image_height = static_cast<int>(cam.image_width / cam.aspect_ratio);
  int num_pixels   = cam.image_width * image_height;
  vec3 * fb;
  hipMallocManaged(&fb, num_pixels * sizeof(vec3));

  camera * dev_cam;
  hipMallocManaged(&dev_cam, sizeof(camera));
  *dev_cam = cam;

  hittable ** dev_list;
  hipMallocManaged(&dev_list, hittable_objects.size() * sizeof(hittable *));
  for (size_t i = 0; i < hittable_objects.size(); ++i) {
    hipMallocManaged(&dev_list[i], sizeof(hittable));
    *dev_list[i] = *hittable_objects[i];
  }

  hittable * dev_world;
  hipMallocManaged(&dev_world, sizeof(hittable_list));
  *(hittable_list *)dev_world = hittable_list(dev_list, hittable_objects.size());

  hiprandState * dev_rand_state;
  hipMallocManaged(&dev_rand_state, num_pixels * sizeof(hiprandState));

  // Kernel launch
  dim3 blocks(cam.image_width / 16, image_height / 16);
  dim3 threads(16, 16);
  render_kernel<<<blocks, threads>>>(fb, cam.image_width, image_height, dev_cam, dev_world,
                                     dev_rand_state);
  hipDeviceSynchronize();

  // Write image to file
  std::ofstream output_ppm_file(output_ppm_path);
  if (!output_ppm_file.is_open()) {
    std::cerr << "Could not open output file: " << output_ppm_path << std::endl;
    return 1;
  }

  output_ppm_file << "P3\n" << cam.image_width << " " << image_height << "\n255\n";
  for (int j = 0; j < image_height; j++) {
    for (int i = 0; i < cam.image_width; i++) {
      int pixel_index = j * cam.image_width + i;
      write_color(output_ppm_file, fb[pixel_index], cam.samples_per_pixel);
    }
  }

  // Free device memory
  hipFree(fb);
  hipFree(dev_cam);
  for (size_t i = 0; i < hittable_objects.size(); ++i) { hipFree(dev_list[i]); }
  hipFree(dev_list);
  hipFree(dev_world);
  hipFree(dev_rand_state);

  for (auto & obj : hittable_objects) { delete obj; }
}
